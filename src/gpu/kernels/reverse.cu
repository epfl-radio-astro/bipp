#include "hip/hip_runtime.h"
#include <algorithm>

#include "bipp/config.h"
#include "gpu/kernels/reverse.hpp"
#include "gpu/util/kernel_launch_grid.hpp"
#include "gpu/util/runtime.hpp"
#include "gpu/util/runtime_api.hpp"

namespace bipp {
namespace gpu {

template <typename T>
__global__ void reverse_1d_kernel(std::size_t n, T* __restrict__ x) {
  for (std::size_t i = threadIdx.x + blockIdx.x * blockDim.x; i < n / 2;
       i += gridDim.x * blockDim.x) {
    T x1 = x[i];
    T x2 = x[n - 1 - i];
    x[n - 1 - i] = x1;
    x[i] = x2;
  }
}

template <typename T>
__global__ void reverse_2d_columns_kernel(std::size_t m, std::size_t n, T* __restrict__ x,
                                          std::size_t ld) {
  int n_2 = n % 2 == 0 ? n / 2 : (n - 1) / 2;
  std::size_t ix = blockIdx.x * blockDim.x + threadIdx.x;
  std::size_t iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix < m && iy < n_2) {
    std::size_t i1 = iy * m + ix;
    std::size_t i2 = (n - 1 - iy) * m + ix;
    T x1 = x[i1];
    T x2 = x[i2];
    x[i2] = x1;
    x[i1] = x2;
  }
}

template <typename T>
auto reverse_1d(Queue& q, std::size_t n, T* x) -> void {
  constexpr int blockSize = 256;
  const dim3 block(std::min<int>(blockSize, q.device_prop().maxThreadsDim[0]), 1, 1);
  const auto grid = kernel_launch_grid(q.device_prop(), {n, 1, 1}, block);
  api::launch_kernel(reverse_1d_kernel<T>, grid, block, 0, q.stream(), n, x);
}

template <typename T>
auto reverse_2d(Queue& q, std::size_t m, std::size_t n, T* x, std::size_t ld) -> void {
  constexpr int blockSize = 256;
  const dim3 block(std::min<int>(blockSize / 8, q.device_prop().maxThreadsDim[0]), 8, 1);
  const auto grid = kernel_launch_grid(q.device_prop(), {m, n, 1}, block);
  api::launch_kernel(reverse_2d_columns_kernel<T>, grid, block, 0, q.stream(), m, n, x, ld);
  hipDeviceSynchronize();
}

template auto reverse_1d<float>(Queue& q, std::size_t n, float* x) -> void;

template auto reverse_1d<double>(Queue& q, std::size_t n, double* x) -> void;

template auto reverse_1d<api::ComplexType<float>>(Queue& q, std::size_t n,
                                                  api::ComplexType<float>* x) -> void;

template auto reverse_1d<api::ComplexType<double>>(Queue& q, std::size_t n,
                                                   api::ComplexType<double>* x) -> void;

template auto reverse_2d<float>(Queue& q, std::size_t m, std::size_t n, float* x, std::size_t ld)
    -> void;

template auto reverse_2d<double>(Queue& q, std::size_t m, std::size_t n, double* x, std::size_t ld)
    -> void;

template auto reverse_2d<api::ComplexType<float>>(Queue& q, std::size_t m, std::size_t n,
                                                  api::ComplexType<float>* x, std::size_t ld)
    -> void;

template auto reverse_2d<api::ComplexType<double>>(Queue& q, std::size_t m, std::size_t n,
                                                   api::ComplexType<double>* x, std::size_t ld)
    -> void;

}  // namespace gpu
}  // namespace bipp
